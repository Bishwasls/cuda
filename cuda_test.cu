#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

// Initialize the RNG state for each thread
__global__ void SetUpRNG(int seed, hiprandState_t* rstates)
{
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  hiprand_init(seed, id, 0, &rstates[id]);
}

int main(int argc, char **argv)
{
  float *ptr_d;
  size_t pitch;
  int nblocks = 4, tpb = 54;
  int nx = 150, ny = 50;

// RNG
  hiprandState_t *rstates;
  hipMalloc(&rstates, nblocks * tpb * sizeof(hiprandState_t));   

  SetUpRNG<<<nblocks, tpb>>>(time(NULL), rstates);

  hipFree(rstates);

  // Device memory
  hipMallocPitch(&ptr_d, &pitch, nx * sizeof(float), ny);

  printf("%s %lu\n", "Pitch is ", pitch);

  hipFree(ptr_d);

  return 0;
}
