/* This program queries the device information using the built-in API functions, and outputs everything to STDOUT */
/* No GPU programming is involved */

#include <stdio.h>

int main(int argc, char **argv)
{
  int device;
  hipDeviceProp_t prop;
   
  device = 0;
  if(argc > 1) device = atoi(argv[1]);
  hipGetDeviceProperties(&prop, device);
  
  printf("%s%s\n", "Device name: ", prop.name);
  printf("%s%d%s%d\n", "Compute capability: ", prop.major, ".", prop.minor);
  printf("%s%d\n", "Number of MPs is ", prop.multiProcessorCount);
  printf("%s%lu%s\n", "Device memmory is ", prop.totalGlobalMem / 1024 / 1024, " MB"); // %lu -- size_t, long unsigned int
  printf("%s%d%s\n", "L2 cache memory is ", prop.l2CacheSize / 1024 / 1024,  " MB");
  printf("%s%d%s%d%s\n", "Maximum number of threads: ", prop.maxThreadsPerBlock, " per block, ",
	 prop.maxThreadsPerMultiProcessor, " per MP");
  printf("%s%d%s%d%s\n", "Register memory: ",  prop.regsPerBlock / 256, " kB per block, ",
	 prop.regsPerMultiprocessor / 256, " kB per MP");
  printf("%s%lu%s%lu%s\n", "Shared memory: ", prop.sharedMemPerBlock / 1024, " kB per block, ",
	 prop.sharedMemPerMultiprocessor / 1024," kB per MP");
  
  printf("%s%d\n", "Single precision perfornamce ratio: ", prop.singleToDoublePrecisionPerfRatio);
  
  return 0;
}
