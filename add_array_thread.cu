#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 64

__global__ void add(int *a, int *b, int *c)
{
  int idx = threadIdx.x;
  if (idx < N)
    {
      c[idx] = a[idx] + b[idx];
    }
}

int main()
{
  int *a, *b, *c;
  int *dev_a, *dev_b, *dev_c;
  
  // Allocate memory space for host a, b, and c
  a = (int *)malloc(N*sizeof(int));
  b = (int *)malloc(N*sizeof(int));
  c = (int *)malloc(N*sizeof(int));
  
  // Allocate memory space for device copies of a, b, and c
  hipMalloc((void **) &dev_a, N*sizeof(int));
  hipMalloc((void **) &dev_b, N*sizeof(int));
  hipMalloc((void **) &dev_c, N*sizeof(int));
  
  // Fill host Arrays
  for (int i = 0; i < N; i++)
    {
      a[i] = i;
      b[i] = 1;
    }

  // Copy from host to device
  hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);

  // Launch the add() kernel on GPU
  add<<<1,N>>>(dev_a, dev_b, dev_c); // 1 block of N threads

  // Copy result back to host
  hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
  
    // Print the result
  for (int i = 0; i < N; i++)
    {
      printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }
  
  return 0;
}
